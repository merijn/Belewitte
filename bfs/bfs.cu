#include <algorithm>
#include "CUDA.hpp"
#include "bfs.hpp"

__device__ unsigned frontier = 0;

void resetFrontier()
{
    const unsigned val = 0;
    CUDA_CHK(hipMemcpyToSymbol(HIP_SYMBOL(frontier), &val, sizeof val));
}

unsigned getFrontier()
{
    unsigned val;
    CUDA_CHK(hipMemcpyFromSymbol(&val, HIP_SYMBOL(frontier), sizeof val));
    return val;
}
