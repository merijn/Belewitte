#include "hip/hip_runtime.h"
#include <algorithm>

#include "bfs.h"
#include "../WarpDispatch.hpp"

#define CUDA_CHK(ans) { cudaAssert((ans), __FILE__, __LINE__); }
void cudaAssert(const hipError_t code, const char *file, const int line);

__device__ bool finished = true;

void resetFinished()
{
    const bool val = true;
    CUDA_CHK(hipMemcpyToSymbol(HIP_SYMBOL(finished), &val, sizeof val));
}

bool getFinished()
{
    bool val;
    CUDA_CHK(hipMemcpyFromSymbol(&val, HIP_SYMBOL(finished), sizeof val));
    return val;
}

__global__ void setArray(int *array, size_t size, int val)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < size) array[idx] = val;
}

template<int W_SZ> __device__ void
memcpy_SIMD(int W_OFF, int cnt, int *dest, int *src)
{
    for (int IDX = W_OFF; IDX < cnt; IDX += W_SZ) {
        dest[IDX] = src[IDX];
    }
    __threadfence_block();
}

template<int W_SZ> __device__ void
expand_bfs_SIMD(int W_OFF, int cnt, int *edges, int *levels, int curr)
{
    for (int IDX = W_OFF; IDX < cnt; IDX += W_SZ) {
        int v = edges[IDX];
        if (levels[v] == -1) {
            levels[v] = curr + 1;
            finished = false;
        }
    }
    __threadfence_block();
}

template<int warp_size, int chunk_size> static __device__ void
warp_bfs_kernel(int N, int curr, int *levels, int *nodes, int *edges)
{
    const int THREAD_ID = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int W_OFF = THREAD_ID % warp_size;
    const int W_ID = THREAD_ID / warp_size;
    extern __shared__ char SMEM[];
    warp_mem_t<chunk_size> *tmp = (warp_mem_t<chunk_size>*)SMEM;
    warp_mem_t<chunk_size> *MY = &tmp[threadIdx.x / warp_size];

    const int v_ = min(W_ID * chunk_size, N);
    const int end = min(chunk_size, (N - v_));

    memcpy_SIMD<warp_size>(W_OFF, end, MY->levels, &levels[v_]);
    memcpy_SIMD<warp_size>(W_OFF, end + 1, MY->nodes, &nodes[v_]);

    for (int v = 0; v < end; v++) {
        if (MY->levels[v] == curr) {
            int num_nbr = MY->nodes[v+1] - MY->nodes[v];
            int *nbrs = &edges[MY->nodes[v]];
            expand_bfs_SIMD<warp_size>(W_OFF, num_nbr, nbrs, levels, curr);
        }
    }
}

template<size_t warp_size, size_t chunk_size> __global__ void
cudabfs(int *nodes, int *edges, int N, int *levels, int curr)
{ warp_bfs_kernel<warp_size, chunk_size>(N, curr, levels, nodes, edges); }

__global__ void set_root(int *depths, int root) { depths[root] = 0; }

template<size_t warp, size_t chunk>
struct BFS {
    static void work()
    { cudabfs<warp, chunk> <<<1, 1, 0 >>>(NULL, NULL, 0, NULL, 0); }
};

void dummyBFS(size_t warp, size_t chunk)
{ warp_dispatch<BFS>::work(warp, chunk); }
