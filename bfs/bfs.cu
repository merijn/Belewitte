#include "hip/hip_runtime.h"
#include <algorithm>
#include "bfs.h"

#define CUDA_CHK(ans) { \
    hipError_t code = ans; \
    if (code != hipSuccess) { \
        cudaAssert(code, __FILE__, __LINE__); \
    } \
}
void __attribute__((noreturn))
cudaAssert(const hipError_t code, const char *file, const int line);

__device__ bool finished = true;

void resetFinished()
{
    const bool val = true;
    CUDA_CHK(hipMemcpyToSymbol(HIP_SYMBOL(finished), &val, sizeof val));
}

bool getFinished()
{
    bool val;
    CUDA_CHK(hipMemcpyFromSymbol(&val, HIP_SYMBOL(finished), sizeof val));
    return val;
}

__global__ void setArray(int *array, size_t size, int val)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < size) array[idx] = val;
}

__global__ void set_root(int *array, int idx)
{ array[idx] = 0; }
