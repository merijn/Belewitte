#include <algorithm>
#include "bfs.hpp"

#define CUDA_CHK(ans) { \
    hipError_t code = ans; \
    if (code != hipSuccess) { \
        cudaAssert(code, __FILE__, __LINE__); \
    } \
}
void __attribute__((noreturn))
cudaAssert(const hipError_t code, const char *file, const int line);

__device__ unsigned frontier = 0;

void resetFrontier()
{
    const unsigned val = 0;
    CUDA_CHK(hipMemcpyToSymbol(HIP_SYMBOL(frontier), &val, sizeof val));
}

unsigned getFrontier()
{
    unsigned val;
    CUDA_CHK(hipMemcpyFromSymbol(&val, HIP_SYMBOL(frontier), sizeof val));
    return val;
}
