#include "hip/hip_runtime.h"
#include "bfs.hpp"

template<typename BFSVariant>
__global__ void
edgeListBfs(EdgeList<unsigned> *graph, int *levels, int depth)
{
    uint64_t startIdx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t size = graph->edge_count;
    BFSVariant bfs;
    int newDepth = depth + 1;


    for (uint64_t idx = startIdx; idx < size; idx += blockDim.x * gridDim.x)
    {
        if (levels[graph->inEdges[idx]] == depth) {
            if (atomicMin(&levels[graph->outEdges[idx]], newDepth) > newDepth) {
                bfs.update();
            }
        }
    }
    bfs.finalise();
}

#ifndef __APPLE__
template __global__ void
edgeListBfs<Reduction<normal>>(EdgeList<unsigned> *, int *, int);

template __global__ void
edgeListBfs<Reduction<bulk>>(EdgeList<unsigned> *, int *, int);

template __global__ void
edgeListBfs<Reduction<warpreduce>>(EdgeList<unsigned> *, int *, int);

template __global__ void
edgeListBfs<Reduction<blockreduce>>(EdgeList<unsigned> *, int *, int);
#endif
