#include "hip/hip_runtime.h"
#include "bfs.h"

__global__ void
edgeListBfs(EdgeListCSR<unsigned,unsigned> *graph, int *levels, int depth)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx < graph->edge_count && levels[graph->inEdges[idx]] == depth) {
        atomicMin(&levels[graph->outEdges[idx]], depth + 1);
        finished = false;
    }
}
