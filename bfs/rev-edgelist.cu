#include "hip/hip_runtime.h"
#include "bfs.hpp"

template<typename BFSVariant>
__global__ void
revEdgeListBfs(EdgeList<unsigned> *graph, int *levels, int depth)
{
    uint64_t size = graph->edge_count;
    BFSVariant bfs;
    int newDepth = depth + 1;

    for (uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
         idx < size && levels[graph->outEdges[idx]] == depth;
         idx += blockDim.x * gridDim.x)
    {
        if (atomicMin(&levels[graph->inEdges[idx]], newDepth) > newDepth) {
            bfs.update();
        }
    }
    bfs.finalise();
}

#ifndef __APPLE__
template __global__ void
revEdgeListBfs<BFS<normal>>(EdgeList<unsigned> *, int *, int);

template __global__ void
revEdgeListBfs<BFS<bulk>>(EdgeList<unsigned> *, int *, int);

template __global__ void
revEdgeListBfs<BFS<warpreduce>>(EdgeList<unsigned> *, int *, int);

template __global__ void
revEdgeListBfs<BFS<blockreduce>>(EdgeList<unsigned> *, int *, int);
#endif
