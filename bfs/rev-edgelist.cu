#include "hip/hip_runtime.h"
#include "bfs.hpp"

template<typename BFSVariant>
__global__ void
revEdgeListBfs(EdgeList<unsigned> *graph, int *levels, int depth)
{
    uint64_t size = graph->edge_count;
    BFSVariant bfs;
    int newDepth = depth + 1;

    for (uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
         idx < size && levels[graph->outEdges[idx]] == depth;
         idx += blockDim.x * gridDim.x)
    {
        if (atomicMin(&levels[graph->inEdges[idx]], newDepth) > newDepth) {
            bfs.update();
        }
    }
    bfs.finalise();
}

#ifndef __APPLE__
template __global__ void
revEdgeListBfs<Reduction<normal>>(EdgeList<unsigned> *, int *, int);

template __global__ void
revEdgeListBfs<Reduction<bulk>>(EdgeList<unsigned> *, int *, int);

template __global__ void
revEdgeListBfs<Reduction<warpreduce>>(EdgeList<unsigned> *, int *, int);

template __global__ void
revEdgeListBfs<Reduction<blockreduce>>(EdgeList<unsigned> *, int *, int);
#endif
