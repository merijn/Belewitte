#include "hip/hip_runtime.h"
#include "bfs.h"

__global__ void
revEdgeListBfs(EdgeList<unsigned> *graph, int *levels, int depth)
{
    uint64_t size = graph->edge_count;
    unsigned count = 1U;
    int newDepth = depth + 1;

    for (uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
         idx < size && levels[graph->outEdges[idx]] == depth;
         idx += blockDim.x * gridDim.x)
    {
        if (atomicMin(&levels[graph->inEdges[idx]], newDepth) > newDepth) {
            updateFrontier(count);
        }
    }
}
