#include "hip/hip_runtime.h"
#include "utils/cuda_utils.hpp"
#include "bfs.hpp"

template<typename BFSVariant>
static __device__ inline int
expand_bfs(BFSVariant &bfs, int W_SZ, unsigned W_OFF, unsigned cnt,
           const unsigned *edges, int *levels, int curr)
{
    int result = 0;
    for (unsigned IDX = W_OFF; IDX < cnt; IDX += W_SZ) {
        if (levels[edges[IDX]] == curr) {
            result = 1;
            break;
        }
    }
    __threadfence_block();
    return __any_sync(0xffffffff, result);
}

template<typename BFSVariant>
__global__ void
vertexPullWarpBfs
( size_t warp_size, size_t chunk_size, CSR<unsigned,unsigned> *graph
, int *levels, int depth)
{
    BFSVariant bfs;
    const size_t vertex_count = graph->vertex_count;
    const int THREAD_ID = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int W_OFF = THREAD_ID % warp_size;
    const size_t W_ID = THREAD_ID / warp_size;
    const size_t BLOCK_W_ID = threadIdx.x / warp_size;

    extern __shared__ int MEM[];
    int *myLevels = &MEM[chunk_size * BLOCK_W_ID];
    unsigned *vertices = (unsigned*) &MEM[(blockDim.x/warp_size) * chunk_size];
    unsigned *myVertices = &vertices[(1+chunk_size) * BLOCK_W_ID];

    const size_t v_ = min(W_ID * chunk_size, vertex_count);
    const size_t end = min(chunk_size, (vertex_count - v_));

    memcpy_SIMD(warp_size, W_OFF, end, myLevels, &levels[v_]);
    memcpy_SIMD(warp_size, W_OFF, end + 1, myVertices, &graph->vertices[v_]);

    bool update;
    for (int v = 0; v < end; v++) {
        const unsigned num_nbr = myVertices[v+1] - myVertices[v];
        const unsigned *nbrs = &graph->edges[myVertices[v]];
        if (myLevels[v] > depth) {
            update = expand_bfs(bfs, warp_size, W_OFF, num_nbr, nbrs, levels, depth);
            if (W_ID == 0 && update) {
                levels[v] = depth + 1;
                bfs.update();
            }
        }
    }
    bfs.finalise();
}

#ifndef __APPLE__
template __global__ void
vertexPullWarpBfs<Reduction<normal>>
(size_t, size_t, CSR<unsigned,unsigned> *, int *, int);

template __global__ void
vertexPullWarpBfs<Reduction<bulk>>
(size_t, size_t, CSR<unsigned,unsigned> *, int *, int);

template __global__ void
vertexPullWarpBfs<Reduction<warpreduce>>
(size_t, size_t, CSR<unsigned,unsigned> *, int *, int);

template __global__ void
vertexPullWarpBfs<Reduction<blockreduce>>
(size_t, size_t, CSR<unsigned,unsigned> *, int *, int);
#endif
