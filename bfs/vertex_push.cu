#include "hip/hip_runtime.h"
#include "bfs.hpp"

template<typename BFSVariant>
__global__ void
vertexPushBfs(CSR<unsigned,unsigned> *graph, int *levels, int depth)
{
    uint64_t size = graph->vertex_count;
    BFSVariant bfs;
    unsigned newDepth = depth + 1;
    for (uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
         idx < size && levels[idx] == depth;
         idx += blockDim.x * gridDim.x)
    {
        unsigned *vertices = graph->vertices;
        unsigned start = vertices[idx];
        unsigned end = vertices[idx + 1];

        for (unsigned i = start; i < end; i++) {
            if (atomicMin(&levels[graph->edges[i]], newDepth) > newDepth) {
                bfs.update();
            }
        }
    }
    bfs.finalise();
}

template __global__ void
vertexPushBfs<BFS<normal>>(CSR<unsigned,unsigned> *, int *, int);

template __global__ void
vertexPushBfs<BFS<bulk>>(CSR<unsigned,unsigned> *, int *, int);

template __global__ void
vertexPushBfs<BFS<warpreduce>>(CSR<unsigned,unsigned> *, int *, int);

template __global__ void
vertexPushBfs<BFS<blockreduce>>(CSR<unsigned,unsigned> *, int *, int);
