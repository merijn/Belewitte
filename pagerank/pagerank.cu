#include "hip/hip_runtime.h"
#include "utils/cuda_utils.hpp"
#include "pagerank.hpp"

__device__ float diff = 0.0;

void resetDiff()
{
    const float val = 0.0;
    CUDA_CHK(hipMemcpyToSymbol(HIP_SYMBOL(diff), &val, sizeof val));
}

float getDiff()
{
    float val;
    CUDA_CHK(hipMemcpyFromSymbol(&val, HIP_SYMBOL(diff), sizeof val));
    return val;
}

static __device__ __forceinline__
void updateDiff(float val)
{
    int lane = threadIdx.x % warpSize;

    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }

    if (lane == 0) atomicAdd(&diff, val);
}

__global__ void
consolidateRank(uint64_t size, float *pagerank, float *new_pagerank, bool)
{
    uint64_t startIdx = (blockIdx.x * blockDim.x) + threadIdx.x;

    for (uint64_t idx = startIdx; idx < size; idx += blockDim.x * gridDim.x) {
        float new_rank = ((1.0 - dampening) / size) + (dampening * new_pagerank[idx]);
        float my_diff = fabsf(new_rank - pagerank[idx]);

        pagerank[idx] = new_rank;
        new_pagerank[idx] = 0.0f;

        updateDiff(my_diff);
    }
}

__global__ void
consolidateRankNoDiv
( InverseVertexCSR<unsigned,unsigned> *graph
, float *pagerank
, float *new_pagerank
, bool notLast
)
{
    uint64_t startIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint64_t vertex_count = graph->vertex_count;

    for (uint64_t idx = startIdx; idx < vertex_count; idx += blockDim.x * gridDim.x) {
        unsigned *outgoing_vertices = &graph->inverse_vertices[idx];

        float new_rank = ((1 - dampening) / vertex_count) + (dampening * new_pagerank[idx]);
        float my_diff = fabsf(new_rank - pagerank[idx]);

        unsigned start = outgoing_vertices[0];
        unsigned end = outgoing_vertices[1];
        unsigned degree = end - start;

        if (degree != 0 && notLast) new_rank = new_rank / degree;
        pagerank[idx] = new_rank;
        new_pagerank[idx] = 0.0f;

        updateDiff(my_diff);
    }
}
