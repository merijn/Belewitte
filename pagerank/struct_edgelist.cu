#include "hip/hip_runtime.h"
#include "pagerank.hpp"

__global__ void
structEdgeListCSR
    ( StructEdgeListCSR<unsigned,unsigned> *graph
    , float *pagerank
    , float *new_pagerank
    )
{
    uint64_t idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx < graph->edge_count) {
        edge<unsigned> *edge = &graph->edges[idx];
        unsigned origin = edge->in;
        unsigned destination = edge->out;
        unsigned *vertices = &graph->vertices[origin];

        unsigned degree = vertices[1] - vertices[0];
        float new_rank = 0.0f;
        if (degree != 0) new_rank = pagerank[origin] / degree;
        atomicAdd(&new_pagerank[destination], new_rank);
    }
}
