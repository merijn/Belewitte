#include "hip/hip_runtime.h"
#include "pagerank.h"

__global__ void vertexPullNoDiv(int *rev_nodes, int *rev_edges, int *nodes, int size, float *pagerank, float *new_pagerank)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    float newRank = 0.0f;

    if (idx < size) {
        for (int i = rev_nodes[idx]; i < rev_nodes[idx + 1]; i++) {
            newRank += pagerank[rev_edges[i]];
        }
    }

    new_pagerank[idx] = newRank;
}
