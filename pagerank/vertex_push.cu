#include "hip/hip_runtime.h"
#include "pagerank.h"

__global__ void
vertexPush
    ( size_t vertex_count
    , size_t edge_count
    , unsigned *nodes
    , unsigned *edges
    , float *pagerank
    , float *new_pagerank)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    int degree;
    float outgoingRank = 0.0f;

    if (idx < vertex_count) {
        degree = nodes[idx + 1] - nodes[idx];

        if (degree != 0) outgoingRank = pagerank[idx] / degree;

        for (int i = nodes[idx]; i < nodes[idx + 1]; i++) {
            atomicAdd(&new_pagerank[edges[i]], outgoingRank);
        }
    }
}
